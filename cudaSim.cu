#include "hip/hip_runtime.h"
// Cuda Tutorial Example
// From: https://devblogs.nvidia.com/even-easier-introduction-cuda/
//


#include <stdlib.h>
#include <stdio.h>
#include <iomanip>
#include <iostream>
#include <sstream>
#include <fstream>
#include <algorithm>
#include <math.h>
#include <random>
#include <vector>
#include <string>
#include <chrono>
#include <map>
#include <unordered_map>

//OpenGL Includes
#ifdef __APPLE__
#include "GLUT/glut.h"
#include <OpenGL/gl.h>
#else
#include <windows.h>
#include "GL/glut.h"
#include <gl/gl.h>
#endif

using namespace std;

//Window diminsions
#define ImageX 1920
#define ImageY 1080
#define ImageZ 1000//1000

#define ImageSize (ImageX * ImageY)

//Grid diminsions
#define GridX 100
#define GridY 100
#define GridZ 100

#define GridSize (GridX * GridY)

#define MAX_UNIT_TIME 200
#define UNIT_TIME 0//16.67		//~60Hz

//-------------------------------Window Variables-------------------------------
int WINDOW_WIDTH = ImageX;
int WINDOW_HEIGHT = ImageY;
int WINDOW_DEPTH = ImageZ;

float Z_NEAR = 0.0001;
float Z_FAR = 1000;

int field_of_view = 90;
int unit_time = UNIT_TIME;
int max_unit_time = MAX_UNIT_TIME;

int fps_delay = 10;
int fps_min = 10000;
int fps_max = 0;
int fps_ct = 0;
int fps;

//-------------------------------Mouse Variables-------------------------------
int g_mouse_x, g_mouse_y, g_mouse_z;
bool mouse_left_pressed = false;
bool mouse_right_pressed = false;


//-------------------------------Draw Variables-------------------------------
bool toggle_simulation = true;
bool swap_buffers = true;

float alive_color[3] = { 1.0, 1.0, 1.0 };
float dead_color[3] = { 0.0, 0.0, 0.0 };


//-------------------------------Timer Struct-------------------------------
struct timer {
	chrono::high_resolution_clock::time_point t_start, t_end;

	void start() { t_start = chrono::high_resolution_clock::now(); }
	void end() { t_end = chrono::high_resolution_clock::now(); }

	float get_time() {
		chrono::duration<float> t = chrono::duration_cast<chrono::duration<float> >(t_end - t_start);
		return t.count();
	}
};

//-------------------------------Utility Variables-------------------------------
timer t;
string info_str = "";


//-------------------------------Object Data-------------------------------

float color[3][3] = { { 0.0, 0.0, 0.0 }, { 1.0, 1.0, 1.0 }, { 0.0, 1.0, 0.0 } };

//Framebuffer used to draw the image
float framebuffer[ImageY][ImageX][3];

//Generation buffers
int *gen_1;
int *gen_2;




//----------------------------------------OpenGL Functions----------------------------------------

int getValue(int *buf, int x, int y);
void set(int *buf, int x, int y, int value);
int getX(int idx);
int getY(int idx);
float getFPS();
void clearFramebuffer();
void setFramebuffer(int x, int y, float R, float G, float B);
int wrap(const int& limit, const int& arg);	//Enables pixel wrapping in the framebuffer. The world now acts as if it were on a sphere.
void setbuffer(bool *buf, int _x, int _y, int _state);
void drawit(void);
void display(void);
void mouseMove(int x, int y);
void mouseClick(int btn, int state, int x, int y);
void passiveMouseMove(int x, int y);
void specialKeyboard(int key, int x, int y);
void keyboard(unsigned char key, int x, int y);
void reshape(int width, int height);
void init(void);


//=========================================================================================================================
//---------------------------------------------------------Cuda------------------------------------------------------------
//=========================================================================================================================


#define NBLOCKS 1
#define NX 24
#define NY 16
#define NTHREAD (NX * NY)


//Convert from 2D to 1D index
__device__ int cuBufIdx(int x, int y)
{
    return (y * ImageX) + x;
}

//Wrap index values
__device__ int cuWrap(int limit, int idx)
{
    return (limit + idx) % limit;
}

// Kernel function to add the elements of two arrays
__global__ void sim( int *buf_1, int *buf_2)
{
	//int id = threadIdx.x + threadIdx.y * NX;
	int id = threadIdx.x + threadIdx.y * NX;
    int x, y, alive_ct;

    //Clear buf_2, prepare it for writing
    for (int i = id; i < ImageSize; i+= NTHREAD)
        buf_2[i] = 0;

    __syncthreads();

    //Write values to buf_2
    for (int i = id; i < ImageSize; i+= NTHREAD)
    {
        x = i % ImageX;
        y = (i - x) / ImageX;
        //idx = (y * ImageX) + x;

        alive_ct = 0;

        alive_ct += buf_1[ cuBufIdx( cuWrap(ImageX - 1, x - 1), cuWrap(ImageY - 1, y - 1) ) ];
        alive_ct += buf_1[ cuBufIdx( cuWrap(ImageX - 1, x    ), cuWrap(ImageY - 1, y - 1) ) ];
        alive_ct += buf_1[ cuBufIdx( cuWrap(ImageX - 1, x + 1), cuWrap(ImageY - 1, y - 1) ) ];
        alive_ct += buf_1[ cuBufIdx( cuWrap(ImageX - 1, x - 1), cuWrap(ImageY - 1, y    ) ) ];
        alive_ct += buf_1[ cuBufIdx( cuWrap(ImageX - 1, x + 1), cuWrap(ImageY - 1, y    ) ) ]; 
        alive_ct += buf_1[ cuBufIdx( cuWrap(ImageX - 1, x - 1), cuWrap(ImageY - 1, y + 1) ) ];
        alive_ct += buf_1[ cuBufIdx( cuWrap(ImageX - 1, x    ), cuWrap(ImageY - 1, y + 1) ) ];
        alive_ct += buf_1[ cuBufIdx( cuWrap(ImageX - 1, x + 1), cuWrap(ImageY - 1, y + 1) ) ];

        //Apply conditions to current cell
        if (buf_1[i])
        {
            if (alive_ct < 2)
            {
                buf_2[i] = false;
                //population_ct--;
            }
            else if (alive_ct == 2 || alive_ct == 3)
            {
                buf_2[i] = true;
            }
            else if (alive_ct > 3)
            {
                buf_2[i] = false;
                //population_ct--;
            }
            else
            {
                buf_2[i] = true;
            }
        }
        else if (alive_ct == 3)
        {
            buf_2[i] = true;
            //population_ct++;
        }

    }
    __syncthreads();

    //Reset buf_1
    for (int i = id; i < ImageSize; i+= NTHREAD)
    {
        buf_1[i] = buf_2[i]; 
    }

	__syncthreads();
}


//=========================================================================================================================
//---------------------------------------------------------Main------------------------------------------------------------
//=========================================================================================================================

int main(int argc, char** argv)
{
    // Allocate Unified Memory – accessible from CPU or GPU
    hipMallocManaged(&gen_1, ImageSize*sizeof(int));
    hipMallocManaged(&gen_2, ImageSize*sizeof(int));

    for (int i = 0; i < ImageSize; i++)
    {
        if(rand()%100 <= 10)
            gen_1[i] = 1;
    }
        

    sim<<<NBLOCKS, dim3(NX, NY)>>>(gen_1, gen_2);

    // Wait for GPU to finish before accessing on host
    hipDeviceSynchronize();

    t.start();
    

//------------------------------------OpenGL-------------------------------------

    //----------------Create Window----------------
	glutInit(&argc, argv);
	glutInitDisplayMode(GLUT_SINGLE | GLUT_RGB);
	glutInitWindowSize(WINDOW_WIDTH, WINDOW_HEIGHT);
	glutInitWindowPosition(100, 100);
	glutCreateWindow("Game of Life");
	init();

    glutMainLoop();
    

    //Clean up memory
    hipFree(gen_1);
    hipFree(gen_2);

    return 0;
}












//=========================================================================================================================
//-------------------------------------------------------OpenGL------------------------------------------------------------
//=========================================================================================================================




//=========================================================================================================================
//-----------------------------------------------------Draw Function-------------------------------------------------------
//=========================================================================================================================

// Draws the scene
void drawit(void)
{
    //Draws the pixel values from the framebuffer
	glDrawPixels(ImageX, ImageY, GL_RGB, GL_FLOAT, framebuffer);
	glFlush();
}

//Draw buffer
void drawBuf(int *buf)
{
    int x, y;
    for (int i = 0; i < ImageSize; i++) {
        x = getX(i);
        y = getY(i);

        framebuffer[y][x][0] = color[buf[i]][0];
        framebuffer[y][x][1] = color[buf[i]][1];
        framebuffer[y][x][2] = color[buf[i]][2];

		//framebuffer[y][x][0] = (buf[i]) ? alive_color[0] : dead_color[0];
		//framebuffer[y][x][1] = (buf[i]) ? alive_color[1] : dead_color[1];
		//framebuffer[y][x][2] = (buf[i]) ? alive_color[2] : dead_color[2];

	}

	drawit();

}


//=========================================================================================================================
//---------------------------------------------------Display Function------------------------------------------------------
//=========================================================================================================================

void display(void)
{
    fps_ct++;

	glClear(GL_COLOR_BUFFER_BIT);

    //info_str = "Generations: " + to_string(generation_ct) + "    Population: " + to_string(population_ct) + "    FPS: " + to_string(fps) + "    Min FPS: " + to_string(fps_min) + "    Draw Size: " + to_string(dot_size);
	info_str = "FPS: " + to_string(fps) + "    Min FPS: " + to_string(fps_min);
    glutSetWindowTitle(info_str.c_str());


    if(toggle_simulation)
    {    
        //Run kernel
        sim<<<NBLOCKS, dim3(NX, NY)>>>(gen_1, gen_2);

        // Wait for GPU to finish before accessing on host
        hipDeviceSynchronize();

        drawBuf(gen_2);

    }


    if(fps_ct % fps_delay)
	{
		t.end();
		fps = (1000 / (t.get_time() * 1000));
		fps_min = ( fps < fps_min ) ? fps : fps_min;
		t.start();
	}

}





int getValue(int *buf, int x, int y)
{
    // id = (y * X_Dim) + x
    
    //  0 1 2
    //  3 4 5
    //  6 7 8

    return buf[ (y * ImageX) + x];
}

void set(int *buf, int x, int y, int value)
{
    buf[(y * ImageX) + x] = value;
}


int getX(int idx)
{
    return idx % ImageX;
}

int getY(int idx)
{
    return (idx - (idx % ImageX)) / ImageX;
}


float getFPS()
{
	return 0.0f;
}



//=========================================================================================================================
//-----------------------------------------------------Clear Frame Buffer--------------------------------------------------
//=========================================================================================================================

// Clears framebuffer to black
void clearFramebuffer()
{
	for (int i = 0; i < ImageSize; i++) {
        framebuffer[ getY(i) ][ getX(i) ][0] = 0.0;
        framebuffer[ getY(i) ][ getX(i) ][1] = 0.0;
        framebuffer[ getY(i) ][ getX(i) ][2] = 0.0;
    }
}

//=========================================================================================================================
//-----------------------------------------------------Set Frame Buffer----------------------------------------------------
//=========================================================================================================================

// Sets pixel x,y to the color RGB
void setFramebuffer(int x, int y, float R, float G, float B)
{
	// changes the origin from the lower-left corner to the upper-left corner
    y = ImageY - 1 - y;
    
    //At this point I am still unsure as to why the 'x' and 'y' must be reversed
    framebuffer[y][x][0] = R;
    framebuffer[y][x][1] = G;
    framebuffer[y][x][2] = B;

}


//Enables pixel wrapping in the framebuffer. The world now acts as if it were on a sphere.
int wrap(const int& limit, const int& arg)
{
	return (limit + arg) % limit;
}


void setbuffer(bool *buf, int _x, int _y, int _state)
{
	int x = wrap(ImageX - 1, _x);
	int y = wrap(ImageY - 1, _y);

    buf[(y * ImageX) + x] = _state;
	
}

//=========================================================================================================================
//----------------------------------------------------Mouse Movement-------------------------------------------------------
//=========================================================================================================================

//Draws when the mouse is clicked and dragged
void mouseMove(int x, int y)
{
	y = ImageY - 1 - y;
	//Compute mouse movement
	//float dx = (x - g_mouse_x);
	//float dy = (y - g_mouse_y);

	//-------------------------Left Mouse Button-------------------------------
	if (mouse_left_pressed) {
		
	}

	//-------------------------Right Mouse Button-------------------------------
	if (mouse_right_pressed) {

	}

	//Update previous mouse position
	g_mouse_x = x;
	g_mouse_y = y;
	
	glutPostRedisplay();
}

//=========================================================================================================================
//-----------------------------------------------------Mouse Click---------------------------------------------------------
//=========================================================================================================================

//Draws when the mouse button is clicked
void mouseClick(int btn, int state, int x, int y)
{
	y = ImageY - 1 - y;
	//Update mouse position
	g_mouse_x = x;
	g_mouse_y = y;

	//-------------------------Left Mouse Button-------------------------------
	if ( btn == GLUT_LEFT_BUTTON ) {

		//Toggle mouse button pressed
		if (state == GLUT_DOWN) 
        {
			mouse_left_pressed = true;

			
		}
		else    //Reset button state
        {	
			mouse_left_pressed = false;

		}
	}

	//-------------------------Right Mouse Button-------------------------------
	if ( btn == GLUT_RIGHT_BUTTON ) 
    {	
		//Toggle mouse button pressed
		if (state == GLUT_DOWN) {
			mouse_right_pressed = true;
		}
		else {	//Reset button state
			mouse_right_pressed = false;
		}
	}

	//-------------------------------------------------------------------------------------------------------------------------

	if (btn == 3) {		//Scroll wheel up
		
	}

	//-------------------------------------------------------------------------------------------------------------------------

	if (btn == 4) {		//Scroll wheel down
		
	}

	//glutPostRedisplay();
}

//=========================================================================================================================
//------------------------------------------------Passive Mouse Function---------------------------------------------------
//=========================================================================================================================

void passiveMouseMove(int x, int y)
{
	y = ImageY - 1 - y;
	//Update mouse position
	g_mouse_x = x;
	g_mouse_y = y;

	//glutPostRedisplay();
}


//=========================================================================================================================
//-------------------------------------------------------------------------------------------------------------------------
//=========================================================================================================================

void specialKeyboard(int key, int x, int y)
{
	switch (key)
	{
		case GLUT_KEY_RIGHT: {
			
			break;
		}
		
		//-------------------------------------------------------------------------------------------------------------------------

		case GLUT_KEY_LEFT: {
			
			break;
		}
		
		//-------------------------------------------------------------------------------------------------------------------------

		case GLUT_KEY_UP: {

			break;
		}

		//-------------------------------------------------------------------------------------------------------------------------

		case GLUT_KEY_DOWN: {	

			break;
		}

		//-------------------------------------------------------------------------------------------------------------------------
		
		case GLUT_KEY_F1: {		//Toggle GameMode

			break;
		}

		//-------------------------------------------------------------------------------------------------------------------------
		
		case GLUT_KEY_F4: {
			exit(0);
			break;
		}
	}

	//glutPostRedisplay();
}

//=========================================================================================================================
//--------------------------------------------------Keyboard Function------------------------------------------------------
//=========================================================================================================================

void keyboard(unsigned char key, int x, int y)
{
	switch (key)
	{
		case '1': {		//Change material | Green
			drawBuf(gen_1);
        
			break;
		}

		case '2': {		//Change material | Blue
			drawBuf(gen_2);

			break;
		}

		case '3': {		//Change material | Red
			
			break;
		}

		

		//---------------------------------Draw Style Options---------------------------------

		case 32: {		
            toggle_simulation = !toggle_simulation;

			break;
		}

		case 'p': {		
			
			break;
		}

		case 'c': {		

			break;
		}

		case 'r': {		//Assign each pixel to a random color

			break;
		}

		//-------------------------------------------------------------------------------------------------------------------------
		
		case '_':
		case '-': {
			
			break;
		}

		//-------------------------------------------------------------------------------------------------------------------------
		
		case '=':
		case '+': {

			break;
		}

		//-------------------------------------------------------------------------------------------------------------------------
	
		default: {
            printf("Key: %c\n", key);
			break;
		}

	}
	//glutPostRedisplay();
}

//=========================================================================================================================
//----------------------------------------------------Reshape Function-----------------------------------------------------
//=========================================================================================================================

void reshape(int width, int height)
{
	WINDOW_WIDTH = width;
	WINDOW_HEIGHT = height;

	glMatrixMode(GL_PROJECTION);
	glLoadIdentity();
	glViewport(0, 0, WINDOW_WIDTH, WINDOW_HEIGHT);
	gluPerspective(field_of_view, float(WINDOW_WIDTH) / WINDOW_HEIGHT, Z_NEAR, Z_FAR);
	glOrtho(-1 * WINDOW_WIDTH / 2.0f, WINDOW_WIDTH / 2.0f, -1 * WINDOW_HEIGHT / 2.0f, WINDOW_HEIGHT / 2.0f, -1 * WINDOW_DEPTH / 2.0f, WINDOW_DEPTH / 2.0f);
	glMatrixMode(GL_MODELVIEW);
}

void updateFrameTimer(int value)
{
	glutPostRedisplay();
	glutTimerFunc(unit_time, updateFrameTimer, 0);
}

//=========================================================================================================================
//-----------------------------------------------------Init Function-------------------------------------------------------
//=========================================================================================================================

void init(void)
{
	
	glClearColor(0.0, 0.0, 0.0, 0.0);
	glMatrixMode(GL_PROJECTION);
	glLoadIdentity();
	glViewport(0, 0, WINDOW_WIDTH, WINDOW_HEIGHT);
	gluPerspective(field_of_view, float(WINDOW_WIDTH) / WINDOW_HEIGHT, Z_NEAR, Z_FAR);
	glOrtho(-1 * WINDOW_WIDTH / 2.0f, WINDOW_WIDTH / 2.0f, -1 * WINDOW_HEIGHT / 2.0f, WINDOW_HEIGHT / 2.0f, -1 * WINDOW_DEPTH / 2.0f, WINDOW_DEPTH / 2.0f);
	glMatrixMode(GL_MODELVIEW);
	

	glutDisplayFunc(display);
	glutKeyboardFunc(keyboard);
	glutSpecialFunc(specialKeyboard);
	glutMotionFunc(mouseMove);
	glutMouseFunc(mouseClick);
	glutPassiveMotionFunc(passiveMouseMove);
	glutReshapeFunc(reshape);
	glutTimerFunc(unit_time, updateFrameTimer, 0);

}
